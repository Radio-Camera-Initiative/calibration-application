#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "calibration.cuh"

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

#define MAXTHREADS 1024
// This is the flagging mask application code for the GPU

/* The assumed shape is as follows:
 *     Visibilities:
 *         (dim1, dim2, polarizations) - float, float
 *     Mask - same as visibilities:
 *         (dim1, dim2, polarizations) - bit
 */

// GPU kernel declaration
/* Thread usage
 *
 * Have a block? for each channel -> the first thread checks if the channel is
 * flagged
 * If it is flagged, then all the threads in the block are each assigned to one
 * visibility in the channel to flag it. Not sure if need to flag all
 * polarizations? Then reassigned until for loop runs out.
 */
__global__ void flag_mask_kernel(
    int dim1,
    int dim2,
    int npol,
    const bool* mask,
    float* vis
) {
    // TODO: store mask or vis in shared memory for quicker access
    int step1 = blockIdx.x * dim2 * npol;

    for (int i = threadIdx.x; i < dim2; i += blockDim.x) { //
        int step2 = (i * npol);
        // each polarization will be set separately

        // use bool to make temp float.
        float m1 = static_cast<float>(!mask[step1 + step2]);
        vis[(step1*CM) + (step2*CM)] *= m1;
        vis[(step1*CM) + (step2*CM) + IM] *= m1;
        float m2 = static_cast<float>(!mask[step1 + step2 + 1]);
        vis[(step1*CM) + (step2*CM) + 2] *= m2;
        vis[(step1*CM) + (step2*CM) + 2 + IM] *= m2;
        float m3 = static_cast<float>(!mask[step1 + step2 + 2]);
        vis[(step1*CM) + (step2*CM) + 4] *= m3;
        vis[(step1*CM) + (step2*CM) + 4 + IM] *= m3;
        float m4 = static_cast<float>(!mask[step1 + step2 + 3]);
        vis[(step1*CM) + (step2*CM) + 6] *= m4;
        vis[(step1*CM) + (step2*CM) + 6 + IM] *= m4;

    }

    return;
}

/*  The assumed shape is as follows:
 *      Visibilities:
 *          (baselines, channels, polarizations) - float, float
 *      Antennas:
 *          (baselines, antennas) - int
 *      Jones:
 *          (antennas, channels, polarizations) - float, float
 */

__device__ void mat_mul_complex(
    float answer[],
    float left[],
    float right[]
) {
    answer[0] = ((left[0] * right[0]) - (left[1] * right[1])) +
        ((left[2] * right[4]) - (left[3] * right[5]));
    answer[1] = ((left[0] * right[1]) + (left[1] * right[0])) +
        ((left[2] * right[5]) + (left[3] * right[4]));

    answer[2] = ((left[0] * right[2]) - (left[1] * right[3])) +
        ((left[2] * right[6]) - (left[3] * right[7]));
    answer[3] = ((left[0] * right[3]) + (left[1] * right[2])) +
        ((left[2] * right[7]) + (left[3] * right[6]));

    answer[4] = ((left[4] * right[0]) - (left[5] * right[1])) +
        ((left[6] * right[4]) - (left[7] * right[5]));
    answer[5] = ((left[4] * right[1]) + (left[5] * right[0])) +
        ((left[6] * right[5]) + (left[7] * right[4]));

    answer[6] = ((left[4] * right[2]) - (left[5] * right[3])) +
        ((left[6] * right[6]) - (left[7] * right[7]));
    answer[7] = ((left[4] * right[3]) + (left[5] * right[2])) +
        ((left[6] * right[7]) + (left[7] * right[6]));
}

__global__ void jones_kernel(
    int nchan,
    int nbaseline,
    int npol,
    int nant,
    float* vis,
    int* ant,
    float* jones
) {
    // TODO: put antennas and/or jones into shared mem for faster access
    int step_size = nchan * npol * CM;
    int base = blockIdx.x;
    
    int ant1 = ant[(blockIdx.x * CM)];
    int ant2 = ant[(blockIdx.x * CM) + 1];

    for (int i = threadIdx.x; i < nchan; i += blockDim.x) {
        int chan = i * npol * CM;
        // [0+1i  2+3i]
        // [4+5i  6+7i]

        // access first matrix for matrixmul mat1 * mat2
        float mat1[8];
        float* matrix = &jones[(ant1 * step_size) + chan];
        for (int j = 0; j < npol * CM; j++) {
            mat1[j] = matrix[j];
        }

        float mat2[8];
        matrix = &vis[(base * step_size) + chan];
        for (int j = 0; j < npol * CM; j++) {
            mat2[j] = matrix[j];
        }
        float mat3[8];

        mat_mul_complex(mat3, mat1, mat2);

        // access second matrix for matrixmul.
        // Also need to conjugate transpose mat1
        matrix = &jones[(ant2 * step_size) + chan];
        for (int j = 0; j < npol * CM; j++) {
            mat1[j] = matrix[j];
        }
        mat1[1] = -mat1[1];
        float temp_re = mat1[2];
        float temp_im = mat1[3];
        mat1[2] = mat1[4];
        mat1[3] = -mat1[5];
        mat1[4] = temp_re;
        mat1[5] = -temp_im;
        mat1[7] = -mat1[7];

        mat_mul_complex(mat2, mat3, mat1);

        // copy mat2 back into visibility
        for (int j = 0; j < npol * CM; j++) {
            vis[(base * step_size) + chan] = mat2[j];
        }// TODO: how to do memcpy for 8 variables?
    }

}


// Make a function to move memory to the GPU, but unneeded with Bifrost as
// everything is already on the GPU
void call_flag_mask_kernel(
    int dim1,
    int dim2,
    int npol,
    const bool* mask,
    float* vis
) {
    std::clog << "dim 1: " << dim1 << "; dim 2: " << dim2 << "; dim 3: " << npol << std::endl;
    assert(dim1 < 65536); // max blocks allowed is 65535
    int size = dim1 * dim2 * npol;

    float* gpu_vis;
    // &gpu_vis gives reference to piece of memory where pointer is stored
    hipMalloc((void**)&gpu_vis, size * CM * sizeof(float));
    cudaCheckErrors("hipMalloc vis fail");
    hipMemcpy(gpu_vis, vis, size * CM * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy vis fail");

    bool* gpu_mask;
    hipMalloc((void**)&gpu_mask, size * sizeof(bool));
    cudaCheckErrors("hipMalloc mask fail");
    hipMemcpy(gpu_mask, mask, size * sizeof(bool), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy mask fail");

    unsigned int blocks = dim1;
    unsigned int threads_per_block = MAXTHREADS;

    std::clog << ">>> Starting kernel" << std::endl;
    flag_mask_kernel<<<blocks, threads_per_block>>> (dim1, dim2, npol, gpu_mask, gpu_vis);

    // Check for errors on kernel call
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
    else
        fprintf(stderr, "No kernel error detected\n");

     hipMemcpy(vis, gpu_vis, size * CM * sizeof(float), hipMemcpyDeviceToHost);

     hipFree(gpu_vis);
     hipFree(gpu_mask);
}

void call_jones_kernel(
    int nchan,
    int nbaseline,
    int npol,
    int nant, 
    float* vis,
    int* ant,
    float* jones
) {
    std::clog << ">>> VIS" << std::endl;
    float* gpu_vis;
    // &gpu_vis gives reference to piece of memory where pointer is stored
    hipMalloc((void**)&gpu_vis, nchan * nbaseline * npol * CM * sizeof(float));
    hipMemcpy(gpu_vis, vis, nchan * nbaseline * npol * CM * sizeof(float), hipMemcpyHostToDevice);

    std::clog << ">>> ANT" << std::endl;
    int* gpu_ant;
    hipMalloc((void**)&gpu_ant, nbaseline * CM * sizeof(int));
    hipMemcpy(gpu_ant, ant, nbaseline * CM * sizeof(int), hipMemcpyHostToDevice);

    std::clog << ">>> JONES" << std::endl;
    float* gpu_jones;
    hipMalloc((void**)&gpu_jones, nchan * nant * npol * CM * sizeof(float));
    hipMemcpy(gpu_jones, jones, nchan * nant * npol * CM * sizeof(float), hipMemcpyHostToDevice);

    unsigned int blocks = nbaseline;
    unsigned int threads_per_block = MAXTHREADS;

    std::clog << ">>> Starting kernel" << std::endl;
    jones_kernel<<<blocks, threads_per_block>>> (nchan, nbaseline, npol, nant, gpu_vis, gpu_ant, gpu_jones);

    // Check for errors on kernel call
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
    else
        fprintf(stderr, "No kernel error detected\n");

     hipMemcpy(vis, gpu_vis, nchan * nbaseline * npol * CM * sizeof(int), hipMemcpyDeviceToHost);

     hipFree(gpu_vis);
     hipFree(gpu_ant);
     hipFree(gpu_jones);
}
